#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>

//////////// Ultility ////////////
#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess) {                                \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                hipGetErrorString(error));                        \
            exit(1);                                               \
        }                                                          \
    }

struct GpuTimer {
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char* fileName, int& numChannels, int& width, int& height, uint8_t*& pixels)
{
    FILE* f = fopen(fileName, "r");
    if (f == NULL) {
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    char type[3];
    fscanf(f, "%s", type);
    if (strcmp(type, "P2") == 0)
        numChannels = 1;
    else if (strcmp(type, "P3") == 0)
        numChannels = 3;
    else // In this exercise, we don't touch other types
    {
        fclose(f);
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    fscanf(f, "%i", &width);
    fscanf(f, "%i", &height);

    int max_val;
    fscanf(f, "%i", &max_val);
    if (max_val > 255) // In this exercise, we assume 1 byte per value
    {
        fclose(f);
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    pixels = (uint8_t*)malloc(width * height * numChannels);
    for (int i = 0; i < width * height * numChannels; i++)
        fscanf(f, "%hhu", &pixels[i]);

    fclose(f);
}

void writePnm(const uint8_t* pixels, int numChannels, int width, int height, char* fileName)
{
    FILE* f = fopen(fileName, "w");
    if (f == NULL) {
        printf("Cannot write %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    if (numChannels == 1)
        fprintf(f, "P2\n");
    else if (numChannels == 3)
        fprintf(f, "P3\n");
    else {
        fclose(f);
        printf("Cannot write %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    fprintf(f, "%i\n%i\n255\n", width, height);

    for (int i = 0; i < width * height * numChannels; i++)
        fprintf(f, "%hhu\n", pixels[i]);

    fclose(f);
}

void writeEnergyMap(const char* filename, const uint32_t* energyMap, int width, int height)
{
    FILE* f = fopen(filename, "w");
    if (f == NULL) {
        printf("Cannot write %s\n", filename);
        exit(EXIT_FAILURE);
    }

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++)
            fprintf(f, "%d ", energyMap[x + y * width]);

        fprintf(f, "\n");
    }

    fclose(f);
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

char* concatStr(const char* s1, const char* s2)
{
    char* result = (char*)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

//////////// Implementations ////////////
#define FILTER_WIDTH 3

const int xSobelFilter[] = {
    1, 0, -1,
    2, 0, -2,
    1, 0, -1
};

const int ySobelFilter[] = {
    1, 2, 1,
    0, 0, 0,
    -1, -2, -1
};

__constant__ int dc_xSobelFilter[FILTER_WIDTH * FILTER_WIDTH];
__constant__ int dc_ySobelFilter[FILTER_WIDTH * FILTER_WIDTH];

/// HOST ///
void hostExtractSeam(const uint32_t* energyMap, int width, int height, uint32_t* outIndexes)
{
    int removeX = 0;
    for (int i = 1; i < width; i++) {
        if (energyMap[removeX] > energyMap[i])
            removeX = i;
    }

    outIndexes[0] = removeX;
    for (int y = 1; y < height; y++) {
        int prevRX = removeX;
        for (int off = -1; off <= 1; off++) {
            int x = min(max(prevRX + off, 0), width - 1);
            if (energyMap[removeX + y * width] > energyMap[x + y * width])
                removeX = x;
        }

        outIndexes[y] = removeX;
    }
}

void hostHighlightSeam(const uint8_t* inPixels, int width, int height, const uint32_t* energyMap, uint8_t* outPixels)
{
    // Find first min energy point
    int removeX = 0;
    for (int i = 1; i < width; i++) {
        if (energyMap[removeX] > energyMap[i])
            removeX = i;
    }

    for (int y = 0; y < height; y++) {
        // Copy pixels
        for (int x = 0; x < width; x++) {
            int xx = x;
            if (xx == removeX) {
                outPixels[(x + y * width) * 3 + 0] = 255;
                outPixels[(x + y * width) * 3 + 1] = 0;
                outPixels[(x + y * width) * 3 + 2] = 0;
                continue;
            }

            for (int channel = 0; channel < 3; channel++)
                outPixels[(x + y * width) * 3 + channel] = inPixels[(xx + y * width) * 3 + channel];
        }

        // Find next min energy point
        if (y < height - 1) {
            int prevRX = removeX;
            for (int off = -1; off <= 1; off++) {
                int xx = min(max(prevRX + off, 0), width - 1);
                if (energyMap[removeX + (y + 1) * width] > energyMap[xx + (y + 1) * width])
                    removeX = xx;
            }
        }
    }
}

/// DEVICE ///

__global__ void deviceGrayscale(const uint8_t* inPixels, int width, int height, uint8_t* gsOutPixels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < width && y < height) {
        int offset = x + y * width;
        gsOutPixels[offset] = uint8_t(
            float(inPixels[offset * 3 + 0]) * 0.299f + float(inPixels[offset * 3 + 1]) * 0.587f + float(inPixels[offset * 3 + 2]) * 0.114f);
    }
}

__global__ void deviceCalcEnergy(const uint8_t* gsInPixels, int width, int height, uint32_t* outEnergy)
{
    extern __shared__ uint8_t s_inPixels[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int blkDimX = blockDim.x;
    int blkDimY = blockDim.y;
    int offsetX = blockIdx.x * blkDimX;
    int offsetY = blockIdx.y * blkDimY;
    int x = tx + offsetX;
    int y = ty + offsetY;

    if (x >= width || y >= height)
        return;

    int padding = FILTER_WIDTH / 2;
    int boundWidth = blkDimX + FILTER_WIDTH - 1;
    int boundHeight = blkDimY + FILTER_WIDTH - 1;

    for (int j = ty; j < boundHeight; j += blkDimY) {
        for (int i = tx; i < boundWidth; i += blkDimX) {
            int xx = min(max(i + offsetX - padding, 0), width - 1);
            int yy = min(max(j + offsetY - padding, 0), height - 1);
            s_inPixels[i + j * boundWidth] = gsInPixels[xx + yy * width];
        }
    }

    __syncthreads();

    int xEdge = 0;
    int yEdge = 0;
    for (int j = 0; j < FILTER_WIDTH; j++) {
        for (int i = 0; i < FILTER_WIDTH; i++) {
            int xx = tx + i;
            int yy = ty + j;
            int filterIdx = i + j * FILTER_WIDTH;

            int pixelVal = s_inPixels[xx + yy * boundWidth];
            xEdge += pixelVal * dc_xSobelFilter[filterIdx];
            yEdge += pixelVal * dc_ySobelFilter[filterIdx];
        }
    }

    xEdge = max(min(xEdge, 255), 0);
    yEdge = max(min(yEdge, 255), 0);
    outEnergy[x + y * width] = xEdge + yEdge;
}

__global__ void deviceRemoveSeam(const uint32_t* removeIndexes, const uint8_t* inPixels, int nChannel, int width, int height, uint8_t* outPixels)
{
    extern __shared__ uint32_t s_removedIdx[];

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= width || y >= height)
        return;

    if (threadIdx.x == 0)
        s_removedIdx[threadIdx.y] = removeIndexes[y];

    __syncthreads();

    int xx = x;
    if (x >= s_removedIdx[threadIdx.y])
        xx++;

    int offDest = (x + y * (width - 1)) * nChannel;
    int offSrc = (xx + y * width) * nChannel;
    for (int i = 0; i < nChannel; i++)
    {
        outPixels[offDest + i] = inPixels[offSrc + i];
    }
}

// FIXME: If image width > blockDim, a.k.a we run on more than 1 grid
//       the seam would never be able to cross between block border
__global__ void deviceFindEnergyMap(const uint32_t* inEnergy, int width, int height, uint32_t* outEnergyMap)
{
    extern __shared__ uint32_t s_rowEnergy[];

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    if (x >= width)
        return;

    int idx = x + (height - 1) * width;
    uint32_t energy = inEnergy[idx];

    s_rowEnergy[threadIdx.x] = energy;
    outEnergyMap[idx] = energy;

    __syncthreads();

    int tx = threadIdx.x;
    for (int y = height - 2; y >= 0; y--) {
        int left = tx == 0 ? INT_MAX : s_rowEnergy[tx - 1];
        int middle = s_rowEnergy[tx];
        int right = INT_MAX;
        if (tx < width - 1 && tx < blockDim.x - 1)
            right = s_rowEnergy[tx + 1];

        idx = x + y * width;
        uint32_t minimum = min(middle, min(left, right));
        uint32_t energy = inEnergy[idx] + minimum;

        __syncthreads();
        s_rowEnergy[tx] = energy;
        outEnergyMap[idx] = energy;
    }
}

uint8_t* seamCarving(
    const uint8_t* inPixels,
    int width, int height,
    int outputWidth,
    dim3 blockSize = dim3(1, 1))
{
    int gsPixelSize = width * height;
    int rgbPixelSize = width * height * 3;
    int energySize = sizeof(uint32_t) * width * height;

    GpuTimer timer;
    timer.Start();

    uint32_t* energyMap = (uint32_t*)malloc(energySize);
    uint32_t* removeIndexes = (uint32_t*)malloc(height * sizeof(uint32_t));
    uint8_t* outPixels = (uint8_t*)malloc(width * height * 3);

    uint8_t* d_gsInpPixels;
    uint8_t* d_rgbInpPixels;
    uint8_t* d_gsOutPixels;
    uint8_t* d_rgbOutPixels;
    uint32_t* d_energy;
    uint32_t* d_energyMap;
    uint32_t* d_removeIndexes;

    CHECK(hipMalloc(&d_gsInpPixels, gsPixelSize));
    CHECK(hipMalloc(&d_rgbInpPixels, rgbPixelSize));
    CHECK(hipMalloc(&d_gsOutPixels, gsPixelSize));
    CHECK(hipMalloc(&d_rgbOutPixels, rgbPixelSize));
    CHECK(hipMalloc(&d_energy, energySize));
    CHECK(hipMalloc(&d_energyMap, energySize));
    CHECK(hipMalloc(&d_removeIndexes, height * sizeof(uint32_t)));

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_xSobelFilter), xSobelFilter, FILTER_WIDTH * FILTER_WIDTH * sizeof(int), 0, hipMemcpyHostToDevice));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_ySobelFilter), ySobelFilter, FILTER_WIDTH * FILTER_WIDTH * sizeof(int), 0, hipMemcpyHostToDevice));

    CHECK(hipMemcpy(d_rgbInpPixels, inPixels, rgbPixelSize, hipMemcpyHostToDevice));

    dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
    deviceGrayscale<<<gridSize, blockSize>>>(d_rgbInpPixels, width, height, d_gsInpPixels);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    
#ifdef WRITE_LOG
    uint8_t* gsInpPixels = (uint8_t*) malloc(gsPixelSize);
    CHECK(hipMemcpy(gsInpPixels, d_gsInpPixels, gsPixelSize, hipMemcpyDeviceToHost));
    writePnm(gsInpPixels, 1, width, height, "grey_dev2.pnm");
#endif

    int carvedWidth = width - outputWidth;
    for (int i = 0; i < carvedWidth; i++) {

        int eWidth = width - i;

        // Calculate energy: E = xSobel + ySobel
        dim3 gridSize((eWidth - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
        int sMemSize = (blockSize.x + FILTER_WIDTH - 1) * (blockSize.y + FILTER_WIDTH - 1) * sizeof(uint8_t);

        deviceCalcEnergy<<<gridSize, blockSize, sMemSize>>>(d_gsInpPixels, eWidth, height, d_energy);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());

#ifdef WRITE_LOG
        CHECK(hipMemcpy(energyMap, d_energy, energySize, hipMemcpyDeviceToHost));
        writeEnergyMap("energy_dev2.txt", energyMap, eWidth, height);
#endif

        // Find energy map line by line
        int fmBlockSize = blockSize.x * blockSize.y;
        int fmGridSize = (eWidth - 1) / fmBlockSize + 1;
        int fmSMemSize = fmBlockSize * sizeof(uint32_t);
        deviceFindEnergyMap<<<fmGridSize, fmBlockSize, fmSMemSize>>>(d_energy, eWidth, height, d_energyMap);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        CHECK(hipMemcpy(energyMap, d_energyMap, energySize, hipMemcpyDeviceToHost));
#ifdef WRITE_LOG
        writeEnergyMap("energy_map_dev2.txt", energyMap, eWidth, height);
        hostHighlightSeam(inPixels, eWidth, height, energyMap, outPixels);
        writePnm(outPixels, 3, eWidth, height, "highlight_dev2.pnm");
#endif
        hostExtractSeam(energyMap, eWidth, height, removeIndexes);
        CHECK(hipMemcpy(d_removeIndexes, removeIndexes, height * sizeof(uint32_t), hipMemcpyHostToDevice));

        int rmSMemSize = blockSize.y * sizeof(uint32_t);

        deviceRemoveSeam<<<gridSize, blockSize, rmSMemSize>>>(d_removeIndexes, d_gsInpPixels, 1, eWidth, height, d_gsOutPixels);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        deviceRemoveSeam<<<gridSize, blockSize, rmSMemSize>>>(d_removeIndexes, d_rgbInpPixels, 3, eWidth, height, d_rgbOutPixels);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        if (i < carvedWidth - 1)
        {
            uint8_t* temp = d_gsInpPixels;
            d_gsInpPixels = d_gsOutPixels;
            d_gsOutPixels = temp;

            temp = d_rgbInpPixels;
            d_rgbInpPixels = d_rgbOutPixels;
            d_rgbOutPixels = temp;
        }
    }

    CHECK(hipMemcpy(outPixels, d_rgbOutPixels, rgbPixelSize, hipMemcpyDeviceToHost));

    free(removeIndexes);
    free(energyMap);

    CHECK(hipFree(d_gsInpPixels));
    CHECK(hipFree(d_rgbInpPixels));
    CHECK(hipFree(d_energy));
    CHECK(hipFree(d_energyMap));

    timer.Stop();
    float time = timer.Elapsed();
    printf("Processing time: %f ms\n\n", time);

    return outPixels;
}

int main(int argc, char** argv)
{
    // printDeviceInfo();

    char* inputFile;
    if (argc >= 2) {
        inputFile = argv[1];
    } else {
        printf("File name is required");
        return 1;
    }

    int outWidth;
    if (argc >= 3) {
        outWidth = atoi(argv[2]);
    } else {
        printf("Output width is required");
        return 1;
    }

    dim3 blockSize(32, 32);
    if (argc >= 4) {
        int v = atoi(argv[3]);
        blockSize.x = v;
        blockSize.y = v;
    }

    int numChannels, width, height;
    uint8_t* inPixels;
    readPnm(inputFile, numChannels, width, height, inPixels);

    if (numChannels != 3) {
        printf("Only RGB image is supported");
        return 1;
    }

    if (outWidth > width - 1) {
        printf("Output width is too big, maximum output width is: %d", width - 1);
        return 1;
    }

    uint8_t* k2OutPixels = seamCarving(inPixels, width, height, outWidth, blockSize);
    char* outFileNameBase = strtok(inputFile, "."); // Get rid of extension
    writePnm(k2OutPixels, 3, outWidth, height, concatStr(outFileNameBase, "_device2.pnm"));
    free(k2OutPixels);

    return 0;
}
